
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

__global__ void kernelDoom(int *a)
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	/*Branch Version*/
	for(int k =0; k < 100;k++)
	{
		if(a[i]>=0)
			a[i] = a[i]+1;
		else
			a[i] = a[i]-1;	
	}
}

__global__ void nobranchDoom(int *a)
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	for(int k = 0; k < 100;k++)
	a[i] = a[i] + (1+2*((a[i] & -1)>>31));
}


#define SAFE_CUDA(call) \
printf("calling: "#call "\n");\
err=call;\
printf(#call ": %d %d\n",err,hipSuccess);

#define TIME_IT(name,x) \
SAFE_CUDA(hipMemcpy(nums_d,o_nums,size,hipMemcpyHostToDevice));\
printf(#x"\n");\
start = clock();\
x;\
SAFE_CUDA(hipMemcpy(nums,nums_d,size,hipMemcpyDeviceToHost));\
end = clock();\
printf("clocks "#name ": %d\n",end-start);\
printf("\n\n");



int main()
{	
	int N = 512*512;
	printf("starting\n");
	size_t size = N*sizeof(int);
	int *o_nums = (int *)malloc(size);
	int *nums = (int *)malloc(size);
	int *nums_d;
	int err = 0;
	int start,end;

	printf("beginning cuda alloc\n");
	SAFE_CUDA(hipMalloc(&nums_d,size));
	for(int i =0;i< N;i++)
		o_nums[i] = i*(1-2*(random()%2));

	TIME_IT(kerneldoom,(kernelDoom<<<512,512>>>(nums_d)));
	
//	TIME_IT(nobranch, (nobranchDoom<<<512,512>>>(nums_d)));

//	TIME_IT(kerneldoom,(kernelDoom<<<512,512>>>(nums_d)));


	hipFree(nums_d);
}
